#include "hip/hip_runtime.h"
/*
This program is free software: you can redistribute it and/or modify
it under the terms of the GNU LESSER GENERAL PUBLIC LICENSE as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU LESSER GENERAL PUBLIC LICENSE for more details.

You should have received a copy of the GNU LESSER GENERAL PUBLIC LICENSE
along with this program.  If not, see <https://www.gnu.org/licenses/>.
*/

#include "DriftCorrection_SRImageXCorr.h"


void SRDriftCorrData_TypeDef_GPU::CorrectSampleShift(string FileName, string oFileName,float PixelSize, int CorrFrameNum, hipStream_t cstream)
{
	// initial information array

	TotalFrame = GetTotalFrame(FileName);


	int cnt = 0;

	CalcGroupNum(TotalFrame, CorrFrameNum);//CorrGroupNum,GroupStartFrame,GroupEndFrame


	GetCorrGroupFluoPos(FileName);//GroupFrameStartPos,GroupFrameEndPos

//	printf("group num:%d %d\n", TotalFrame, CorrGroupNum);

	/*
	for (cnt; cnt < CorrGroupNum; cnt++)
	{
	printf("frame pos: %d-%d, %d-%d\n", GroupStartFrame[cnt], GroupEndFrame[cnt], GroupFrameStartPos[cnt], GroupFrameEndPos[cnt]);
	}
	*/

	float ShiftX = 0;
	float ShiftY = 0;
	int CorrShiftBiasX = 0;
	int CorrShiftBiasY = 0;

	// rend  super-resolution image
	// 0th is the reference image and the others are drifted image


	for (cnt = 0; cnt < CorrGroupNum; cnt += 1)
	{
		RenderSlice(FileName, cnt, PixelSize, cstream);

		if (cnt > 0)
		{
			GetSliceShift( &ShiftX, &ShiftY, CorrShiftBiasX, CorrShiftBiasY, cstream);
			CorrShiftBiasX = (int)ShiftX;
			CorrShiftBiasY = (int)ShiftY;
		}

		XSliceShift[cnt] = ShiftX / SRShiftCorr_PixelZoom;
		YSliceShift[cnt] = ShiftY / SRShiftCorr_PixelZoom;

		printf("x y shift(pixel):%f %f\n", XSliceShift[cnt], YSliceShift[cnt]);

	}

	ShiftInterpolation();


	ApplyShiftTop(FileName, oFileName, BIT0 | BIT1, cstream);


	hipStreamSynchronize(cstream);

}

void HandleErr(hipError_t err, const char* str)
{
	if (err != hipSuccess)
	{
		printf("cuda err:%s, %s\n", str, hipGetErrorString(err));
	}
	else
	{
		printf("cuda suc:%s\n", str);
	}
}

void SRDriftCorrData_TypeDef_GPU::Init(int RawImgWidth, int RawImgHigh, int MaxFrameNum)
{

	hipError_t err;

	// image information
	ImageWidth = RawImgWidth;
	ImageHigh = RawImgHigh;

	SRImageWidth = RawImgWidth*SRShiftCorr_PixelZoom;
	SRImageHigh = RawImgHigh*SRShiftCorr_PixelZoom;
		

	// create inf array for each group
	GroupStartFrame = new int[MaxCorrGroupNum]; // start frame of each group
	GroupEndFrame = new int[MaxCorrGroupNum]; // start frame of each group

	GroupFrameStartPos = new int[MaxCorrGroupNum]; // start fluo position of each group
	GroupFrameEndPos = new int[MaxCorrGroupNum]; // start fluo position of each group



	XSliceShift = new float[MaxCorrGroupNum];
	YSliceShift = new float[MaxCorrGroupNum];


	hipHostMalloc((void **)&XFrameShift, MaxFrameNum*sizeof(float));
	hipHostMalloc((void **)&YFrameShift, MaxFrameNum*sizeof(float));

	hipMalloc((void **)&d_XFrameShift, MaxFrameNum*sizeof(float));
	hipMalloc((void **)&d_YFrameShift, MaxFrameNum*sizeof(float));

	hipHostMalloc((void **)&h_LocArry, PointNumTh * 2 * OutParaNumGS2D*sizeof(float));
	hipMalloc((void **)&d_LocArry, PointNumTh * 2 * OutParaNumGS2D*sizeof(float));

	err = hipHostMalloc((void **)&h_FillImg1, SRImageWidth*SRImageHigh*sizeof(float));
	HandleErr(err, "hipHostMalloc shift corr h_FillImg1");
	err = hipHostMalloc((void **)&h_FillImg2, SRImageWidth*SRImageHigh*sizeof(float));
	HandleErr(err, "hipHostMalloc shift corr h_FillImg2");
	err = hipHostMalloc((void **)&h_SumLine, SRImageWidth*sizeof(float));


	err = hipMalloc((void **)&d_FillImg1, SRImageWidth*SRImageHigh*sizeof(float));
	HandleErr(err, "hipMalloc shift corr d_FillImg1");
	err = hipMalloc((void **)&d_FillImg2, SRImageWidth*SRImageHigh*sizeof(float));
	HandleErr(err, "hipMalloc shift corr d_FillImg2");
	err = hipMalloc((void **)&d_MulImg, SRImageWidth*SRImageHigh*sizeof(float));
	HandleErr(err, "hipMalloc shift corr d_MulImg");
	hipMalloc((void **)&d_SumLine, SRImageWidth*sizeof(float));


}


void SRDriftCorrData_TypeDef_GPU::Deinit()
{
	hipError_t err;

	delete[] GroupStartFrame;
	delete[] GroupEndFrame;

	delete[] GroupFrameStartPos;
	delete[] GroupFrameEndPos;


	delete[] XSliceShift;
	delete[] YSliceShift;

	hipHostFree(XFrameShift);
	hipHostFree(YFrameShift);

	hipFree(d_XFrameShift);
	hipFree(d_YFrameShift);


	hipHostFree(h_LocArry);
	hipFree(d_LocArry);

	err = hipHostFree(h_FillImg1);
	HandleErr(err, "hipHostFree shift corr h_FillImg1");
	err = hipHostFree(h_FillImg2);
	HandleErr(err, "hipHostFree shift corr h_FillImg2");
	err = hipHostFree(h_SumLine);

	err = hipFree(d_FillImg1);
	HandleErr(err, "hipFree shift corr d_FillImg1");
	err = hipFree(d_FillImg2);
	HandleErr(err, "hipFree shift corr d_FillImg2");
	err = hipFree(d_MulImg);
	err = hipFree(d_SumLine);

}


void SRDriftCorrData_TypeDef_GPU::ResetFillImage(float *d_SRIntensityImg, int SRImageWidth, int SRImageHigh, hipStream_t cstream)
{
	
	hipMemsetAsync(d_SRIntensityImg, 0, SRImageWidth*SRImageHigh*sizeof(float));
	hipStreamSynchronize(cstream);

}


void SRDriftCorrData_TypeDef_GPU::GetSliceShift(float *ShiftX, float *ShiftY, int CorrShiftBiasX, int CorrShiftBiasY, hipStream_t cstream)
{
	// calculate cross correletion
	// don't calculate all, only calculate a useful region

	int CorrShiftX = 0;
	int CorrShiftY = 0;


	//	printf("cor size:%d %d\n", SRImageWidth, SRImageHigh);

	int cnt = 0;
	double CorrResult[CorrSize][CorrSize];

	int xcnt, ycnt;

	double MaxSumDat = 0;
	int MaxPosX = 0, MaxPosY = 0;


	float SumCorrX[2 * FittingRadius + 1]; // sum along x,y direction for fitting region
	float SumCorrY[2 * FittingRadius + 1]; // sum along x,y direction for fitting region

	int FitXS, FitXE, FitYS, FitYE;// X,Y start,end pos


	// cross-correlation calculation for a 51x51 region
	for (ycnt = 0; ycnt < CorrSize; ycnt++)
	{
		for (xcnt = 0; xcnt < CorrSize; xcnt++)
		{
			CorrShiftX = xcnt - HalfCorrSize;
			CorrShiftY = ycnt - HalfCorrSize;

			CorrResult[ycnt][xcnt] = CrossCorrelation(CorrShiftX, CorrShiftY, CorrShiftBiasX, CorrShiftBiasY, cstream);
			if (MaxSumDat < CorrResult[ycnt][xcnt])
			{
				MaxSumDat = CorrResult[ycnt][xcnt];
				MaxPosX = xcnt;
				MaxPosY = ycnt;
			}
			//			printf("%.1f ", CorrResult[ycnt][xcnt]);
		}
	}
	// normalize the correlation result
	for (ycnt = 0; ycnt < CorrSize; ycnt++)
	{
		for (xcnt = 0; xcnt < CorrSize; xcnt++)
		{

			CorrResult[ycnt][xcnt] = CorrResult[ycnt][xcnt] / MaxSumDat * 1000;
			//			printf("%.1f ", CorrResult[ycnt][xcnt]);
		}
	}


	//	printf("\nMaxPosX:%d %d\n", MaxPosX, MaxPosY);

	// find the fitting region centered with the max pos
	FitXS = max(MaxPosX - FittingRadius, 0);
	FitYS = max(MaxPosY - FittingRadius, 0);
	FitXE = min(MaxPosX + FittingRadius, CorrSize - 1);
	FitYE = min(MaxPosY + FittingRadius, CorrSize - 1);

	int xhlen = (FitXE - FitXS + 1) / 2;
	int yhlen = (FitYE - FitYS + 1) / 2;

	for (cnt = 0; cnt < 2 * FittingRadius + 1; cnt++)
	{
		SumCorrX[cnt] = 0;
		SumCorrY[cnt] = 0;
	}
	// center of mass fitting of gaussian shape cross-correlation
	for (ycnt = FitYS; ycnt <= FitYE; ycnt++)
	{
		for (xcnt = FitXS; xcnt <= FitXE; xcnt++)
		{
			SumCorrX[FittingRadius - xhlen + (xcnt - FitXS)] += CorrResult[ycnt][xcnt];
			SumCorrY[FittingRadius - yhlen + (ycnt - FitYS)] += CorrResult[ycnt][xcnt];
		}
	}

	float wSum = 0, cSum = 0; // weighted sum for center of mass
	float CenterX, CenterY;

	for (cnt = 0; cnt < 2 * FittingRadius + 1; cnt++)
	{
		if (SumCorrX[cnt] == 0)continue;

		wSum += SumCorrX[cnt] * (cnt + 1);
		cSum += SumCorrX[cnt];

	}
	CenterX = wSum / cSum - (FittingRadius + 1);

	wSum = 0;
	cSum = 0;

	for (cnt = 0; cnt < 2 * FittingRadius + 1; cnt++)
	{
		if (SumCorrY[cnt] == 0)continue;

		wSum += SumCorrY[cnt] * (cnt + 1);
		cSum += SumCorrY[cnt];

	}
	CenterY = wSum / cSum - (FittingRadius + 1);


	*ShiftX = CorrShiftBiasX + MaxPosX - HalfCorrSize + CenterX;
	*ShiftY = CorrShiftBiasY + MaxPosY - HalfCorrSize + CenterY;

	//	printf("shift pos:%f %f\n", *ShiftX, *ShiftY);

}

